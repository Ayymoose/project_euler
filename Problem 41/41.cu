#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 512
#define GRID_SIZE 1024
#define N 500000000//(GRID_SIZE*GRID_SIZE*BLOCK_SIZE)

#define checkCudaErrors(error) {\
	if (error != hipSuccess) {\
		printf("CUDA Error - %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));\
		exit(1);\
		}\
}\

__global__ void init_sieve(int *primes) {
	int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; //(blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y) + threadIdx.x;
	if (i<N) {
		primes[i] = i;
	}
}

__global__ void prime_sieve(int *primes) {
	
	//unsigned long long i = threadIdx.x + blockIdx.x * blockDim.x;
   	int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; //(blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y) + threadIdx.x;
	if (i > 1 && i<N/2) {	
		for (int j=2*i; j<N; j+=i) {
			primes[j]=0;
		}

	}
}
__global__ void prime_count(int *primes) {
	int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if (i > 1 && i<N) {
		if (primes[i]) {
			atomicAdd(&primes[0],1);
		}
	}
}

int main() {
	int *h_primes=(int*)malloc(N * sizeof(int));
	if (h_primes == NULL) {
		printf("malloc failed\n");
		exit(1);
	}
	int *d_primes;
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(GRID_SIZE,GRID_SIZE);

	checkCudaErrors(hipMalloc( (void**)&d_primes, N*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_primes,h_primes,N*sizeof(int),hipMemcpyHostToDevice));	
	init_sieve<<<dimGrid,dimBlock>>>(d_primes);
	prime_sieve<<<dimGrid,dimBlock>>>(d_primes);
	prime_count<<<dimGrid,dimBlock>>>(d_primes);
	checkCudaErrors(hipMemcpy(h_primes,d_primes,N*sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_primes));
	
	int size = 0;
	int total = N;
	size = h_primes[0];

	printf("Length = %d\tPrimes = %d\n",total,size);

	free(h_primes);
	return 0;

	// GTX 780 Ti ~3GB 
	// nvcc 41.cu -o 41.o -arch=sm_30 -Xcompiler -Ofast
}

/*
	100m 0m15.093s
	299m 0m59.660s
	399m 1m27.943s
	499m 1m57.786s
	599m 2m35.930s
	699m 3m23.022s
	799m 4m14.884s
	899m 5m20.725s

4	999m 6m26.204s
12	999m 1m47.437s

*/
