#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
	#include "euler.h"
}

#define BLOCK_SIZE 1024
#define GRID_SIZE 1024
#define N 1000000000//(GRID_SIZE*GRID_SIZE*BLOCK_SIZE)

#define checkCudaErrors(error) {\
	if (error != hipSuccess) {\
		printf("CUDA Error - %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));\
		exit(1);\
		}\
}\

__global__ void init_sieve(int *primes) {
	int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; 
	if (i<N) {
		primes[i] = i;
	}
}

__global__ void prime_sieve(int *primes) {
   	int i = ((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x); 
	if (i > 1 && i<N/2) {	
		for (int j=2*i; j<N; j+=i) {
			primes[j]=0;
		}

	}
}

int main() {
	int *h_primes=(int*)malloc(N * sizeof(int));
	if (h_primes == NULL) {
		printf("malloc failed\n");
		exit(1);
	}
	int *d_primes;
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(GRID_SIZE,GRID_SIZE);

	checkCudaErrors(hipMalloc( (void**)&d_primes, N*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_primes,h_primes,N*sizeof(int),hipMemcpyHostToDevice));	
	init_sieve<<<dimGrid,dimBlock>>>(d_primes);
	prime_sieve<<<dimGrid,dimBlock>>>(d_primes);
	checkCudaErrors(hipMemcpy(h_primes,d_primes,N*sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_primes));
	
	int index=N-1;
	while (index) {
		if (h_primes[index]) {
			if (is_pandigital(h_primes[index])) {
				printf("%d\n",h_primes[index]);
				break;
			}
		}
		index--;
	}
	
	free(h_primes);
	return 0;

}
