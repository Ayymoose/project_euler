#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 512
#define THREAD_SIZE 512 
#define L 150000

#define checkCudaErrors(error) {\
	if (error != hipSuccess) {\
		printf("CUDA Error - %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));\
		exit(1);\
		}\
}\

__global__ void singular_integer_triangle(int *array) {
	int p = blockIdx.x * blockDim.x + threadIdx.x;		
	if (p <= L) {
	for (long a=sqrt((float)p); a<p/2; a++) {
		for (long b=a+1; b<p/2; b++) {
			if (a*a + b*b == (p-a-b)*(p-a-b)) {
				array[p]++;
				if (array[p] > 1) return;
			}
		}
	}
	}

}

int main() {
	int *h_array = (int*)calloc(L,sizeof(int));
	if (h_array == NULL) {
		printf("malloc() failed/n");
		exit(1);
	}
	int *d_array;
	checkCudaErrors(hipMalloc( (void**)&d_array, L*sizeof(int)));
        checkCudaErrors(hipMemcpy(d_array,h_array,L*sizeof(int),hipMemcpyHostToDevice));	
	singular_integer_triangle<<<BLOCK_SIZE,THREAD_SIZE>>>(d_array);
	checkCudaErrors(hipMemcpy(h_array,d_array,L*sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_array));

	int count = 0;
	//   printf("[");
	for (int i=0; i<L; i++) {
		if (h_array[i]==1) count++;
	//	printf("%d,",h_array[i]);
	}
	//printf("%d]\n",h_array[29]);*/
	printf("%d\n",count);

	free(h_array);
	return 0;
}
